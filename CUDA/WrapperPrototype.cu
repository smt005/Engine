#include "hip/hip_runtime.h"

#include  "WrapperPrototype.h"
#include "Wrapper.h"
#include <thread>
#include <vector>

#if ENABLE_CUDA

#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>

namespace {
    void GetForceCPU(int count, int offset, float* masses, float* positionsX, float* positionsY, float* forcesX, float* forcesY, int threadId) {
        double _constGravity = 0.01f;
        int statIndex = threadId * offset;
        int endIndex = statIndex + offset;
        if (endIndex > count) {
            endIndex = count;
        }

        int sizeData = count;

        for (int index = statIndex; index < endIndex; ++index) {
            for (size_t otherIndex = 0; otherIndex < sizeData; ++otherIndex) {
                if (index == otherIndex) {
                    continue;
                }

                float gravityVecX = positionsX[otherIndex] - positionsX[index];
                float gravityVecY = positionsY[otherIndex] - positionsY[index];

                double dist = sqrt(gravityVecX * gravityVecX + gravityVecY * gravityVecY);
                gravityVecX /= dist;
                gravityVecY /= dist;

                double force = _constGravity * (masses[index] * masses[otherIndex]) / (dist * dist);
                gravityVecX *= force;
                gravityVecY *= force;

                forcesX[index] += gravityVecX;
                forcesY[index] += gravityVecY;
            }
        }
    }

    __global__
    void GetForceGPU(int* count, int* offset, float* masses, float* positionsX, float* positionsY, float* forcesX, float* forcesY) {
        double _constGravity = 0.01f;
        int statIndex = blockIdx.x * *offset;
        int endIndex = statIndex + *offset;
        if (endIndex > *count) {
            endIndex = *count;
        }

        int sizeData = *count;

        for (int index = statIndex; index < endIndex; ++index) {
            for (size_t otherIndex = 0; otherIndex < sizeData; ++otherIndex) {
                if (index == otherIndex) {
                    continue;
                }

                float gravityVecX = positionsX[otherIndex] - positionsX[index];
                float gravityVecY = positionsY[otherIndex] - positionsY[index];

                double dist = sqrt(gravityVecX * gravityVecX + gravityVecY * gravityVecY);
                gravityVecX /= dist;
                gravityVecY /= dist;

                double force = _constGravity * (masses[index] * masses[otherIndex]) / (dist * dist);
                gravityVecX *= force;
                gravityVecY *= force;

                forcesX[index] += gravityVecX;
                forcesY[index] += gravityVecY;
            }
        }
    }
}

void CUDA_Prototype::GetForcesCPUStatic(int count, float* masses, float* positionsX, float* positionsY, float* forcesX, float* forcesY) {
    unsigned int counThread = static_cast<double>(std::thread::hardware_concurrency());
    unsigned int sizeB = count;

    if ((sizeB * 2) > counThread) {
        int offst = sizeB / counThread;
        if ((sizeB % counThread) > 0) {
            ++offst;
        }

        std::vector<std::thread> threads;
        threads.reserve(counThread);

        for (unsigned int threadId = 0; threadId < counThread; ++threadId) {
            threads.emplace_back([&]() {
                GetForceCPU(count, offst, masses, positionsX, positionsY, forcesX, forcesY, threadId);
            });
        }

        for (std::thread& th : threads) {
            th.join();
        }
    }
    else
    {
        GetForceCPU(count, count, masses, positionsX, positionsY, forcesX, forcesY, 0);
    }
}

void CUDA_Prototype::GetForcesGPUStatic(int count, float* masses, float* positionsX, float* positionsY, float* forcesX, float* forcesY) {
    unsigned int counThread = CUDA::warpSize;
    if (counThread == 0) {
        return;
    }

    int offst = count / counThread;
    if ((count % counThread) > 0) {
        ++offst;
    }

    //...
    int* devCount;
    int* devOffset;
    float* devMasses;
    float* devPositionsX;
    float* devPositionsY;
    float* devForcesX;
    float* devForcesY;

    hipMalloc(&devCount,               sizeof(float));
    hipMalloc(&devOffset,              sizeof(float));
    hipMalloc(&devMasses,      count * sizeof(float));
    hipMalloc(&devPositionsX,  count * sizeof(float));
    hipMalloc(&devPositionsY,  count * sizeof(float));
    hipMalloc(&devForcesX,     count * sizeof(float));
    hipMalloc(&devForcesY,     count * sizeof(float));

    hipMemcpy(devCount,        &count,     count * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devOffset,       &offst,     count * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devMasses,       masses,     count * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devPositionsX,   positionsX, count * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devPositionsY,   positionsY, count * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devForcesX,      forcesX,    count * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devForcesY,      forcesY,    count * sizeof(float), hipMemcpyHostToDevice);

    GetForceGPU <<<counThread, 1>>> (devCount, devOffset, devMasses, devPositionsX, devPositionsY, devForcesX, devForcesY);

    hipMemcpy(devForcesX, forcesX, count * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(devForcesY, forcesY, count * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(devCount);
    hipFree(devOffset);
    hipFree(devMasses);
    hipFree(positionsX);
    hipFree(positionsY);
    hipFree(devForcesX);
    hipFree(devForcesY);
}

//...
#else
    void testCUDA(void) {}

    void CUDA::GetProperty() {}
    void CUDA::PrintInfo() {}
#endif
