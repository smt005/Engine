#include "hip/hip_runtime.h"

#include  "WrapperPrototype.h"
#include "Wrapper.h"
#include <thread>
#include <vector>
#include <iostream>

#if ENABLE_CUDA

#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>

namespace {
    void GetForceCPU(int *count, int *offset, float* masses, float* positionsX, float* positionsY, float* forcesX, float* forcesY, int threadId) {
        double _constGravity = 0.01f;
        int statIndex = threadId * *offset;
        int endIndex = statIndex +* offset;
        if (endIndex > *count) {
            endIndex = *count;
        }

        int sizeData = *count;
        float gravityVecX;
        float gravityVecY;
        double dist;
        double force;

        for (int index = statIndex; index < endIndex; ++index) {
            float& fX = forcesX[index];
            float& fY = forcesY[index];

            for (size_t otherIndex = 0; otherIndex < sizeData; ++otherIndex) {
                if (index == otherIndex) {
                    continue;
                }

                gravityVecX = positionsX[otherIndex] - positionsX[index];
                gravityVecY = positionsY[otherIndex] - positionsY[index];

                dist = sqrt(gravityVecX * gravityVecX + gravityVecY * gravityVecY);
                gravityVecX /= dist;
                gravityVecY /= dist;

                force = _constGravity * (masses[index] * masses[otherIndex]) / (dist * dist);
                gravityVecX *= force;
                gravityVecY *= force;

                fX += gravityVecX;
                fY += gravityVecY;
            }
        }
    }
}

    void CUDA_Prototype::GetForcesCPUStatic(int count, float* masses, float* positionsX, float* positionsY, float* forcesX, float* forcesY) {
        unsigned int counThread = static_cast<double>(std::thread::hardware_concurrency());
        unsigned int sizeB = count;

        if ((sizeB * 2) > counThread) {
            int offst = sizeB / counThread;
            if ((sizeB % counThread) > 0) {
                ++offst;
            }

            std::vector<std::thread> threads;
            threads.reserve(counThread);

            for (unsigned int threadId = 0; threadId < counThread; ++threadId) {
                threads.emplace_back([&]() {
                    GetForceCPU(&count, &offst, masses, positionsX, positionsY, forcesX, forcesY, threadId);
                    });
            }

            for (std::thread& th : threads) {
                th.join();
            }
        }
        else
        {
            GetForceCPU(&count, &count, masses, positionsX, positionsY, forcesX, forcesY, 0);
        }
    }

    //////////////////////////////////////////////////////////////////////////////////////////////////////////
    // GPU ///////////////////////////////////////////////////////////////////////////////////////////////////
    //////////////////////////////////////////////////////////////////////////////////////////////////////////
    
namespace {
    __global__
    void GetForceGPU(int* count, int* offset, float* masses, float* positionsX, float* positionsY, float* forcesX, float* forcesY) {
        double _constGravity = 0.01f;
        int statIndex = (threadIdx.x + blockIdx.x * blockDim.x) * *offset;
        int endIndex = statIndex + *offset;
        if (endIndex > *count) {
            endIndex = *count;
        }

        int sizeData = *count;
        float gravityVecX = 0;
        float gravityVecY = 0;
        double dist;
        double force;

        for (int index = statIndex; index < endIndex; ++index) {
            float& fX = forcesX[index];
            float& fY = forcesY[index];

            for (size_t otherIndex = 0; otherIndex < sizeData; ++otherIndex) {
                if (index == otherIndex) {
                    continue;
                }

                gravityVecX = positionsX[otherIndex] - positionsX[index];
                gravityVecY = positionsY[otherIndex] - positionsY[index];

                dist = sqrt(gravityVecX * gravityVecX + gravityVecY * gravityVecY);
                gravityVecX /= dist;
                gravityVecY /= dist;

                force = _constGravity * (masses[index] * masses[otherIndex]) / (dist * dist);
                gravityVecX *= force;
                gravityVecY *= force;

                fX += gravityVecX;
                fY += gravityVecY;
            }
        }
    }
}

void CUDA_Prototype::GetForcesGPUStatic(int count, float* masses, float* positionsX, float* positionsY, float* forcesX, float* forcesY) {
    unsigned int counThread = count < CUDA::maxThreadsPerBlock ? count : CUDA::maxThreadsPerBlock;

    unsigned int countBlock = (count + counThread - 1) / counThread;
    countBlock = countBlock > CUDA::maxGridSize[1] ? CUDA::maxGridSize[1] : countBlock;

    int offset = count / (counThread * countBlock);
    if ((count % (counThread * countBlock)) > 0) {
        ++offset;
    }

    //...
    int* devCount;
    int* devOffset;
    float* devMasses;
    float* devPositionsX;
    float* devPositionsY;
    float* devForcesX;
    float* devForcesY;

    hipMalloc(&devCount,               sizeof(int));
    hipMalloc(&devOffset,              sizeof(int));
    hipMalloc(&devMasses,      count * sizeof(float));
    hipMalloc(&devPositionsX,  count * sizeof(float));
    hipMalloc(&devPositionsY,  count * sizeof(float));
    hipMalloc(&devForcesX,     count * sizeof(float));
    hipMalloc(&devForcesY,     count * sizeof(float));

    hipMemcpy(devCount,        &count,             sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(devOffset,       &offset,             sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(devMasses,       masses,     count * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devPositionsX,   positionsX, count * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devPositionsY,   positionsY, count * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devForcesX,      forcesX,    count * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devForcesY,      forcesY,    count * sizeof(float), hipMemcpyHostToDevice);

    GetForceGPU <<<countBlock, counThread>>> (devCount, devOffset, devMasses, devPositionsX, devPositionsY, devForcesX, devForcesY);

    hipMemcpy(forcesX, devForcesX, count * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(forcesY, devForcesY, count * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(devCount);
    hipFree(devOffset);
    hipFree(devMasses);
    hipFree(positionsX);
    hipFree(positionsY);
    hipFree(devForcesX);
    hipFree(devForcesY);
}

//...
#else
    void testCUDA(void) {}

    void CUDA::GetProperty() {}
    void CUDA::PrintInfo() {}
#endif
