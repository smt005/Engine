#include "hip/hip_runtime.h"

#include "Wrapper.h"
#include <thread>
#include <vector>

int CUDA::deviceCount = -1;
int CUDA::warpSize = 0;

#if ENABLE_CUDA

#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>

__global__
void saxpy(int n, float a, float* x, float* y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) y[i] = a * x[i] + y[i];
}

void testCUDA(void)
{
    int N = 1 << 20;
    float* x, * y, * d_x, * d_y;
    x = (float*)malloc(N * sizeof(float));
    y = (float*)malloc(N * sizeof(float));

    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));

    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

    // Perform SAXPY on 1M elements
    saxpy << <(N + 255) / 256, 256 >> > (N, 2.0f, d_x, d_y);

    hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = std::max(maxError, std::abs(y[i] - 4.0f));
    printf("Max error: %f\n", maxError);

    hipFree(d_x);
    hipFree(d_y);
    free(x);
    free(y);
}

void CUDA::GetProperty() {
    if (deviceCount == -1) {
        hipGetDeviceCount(&deviceCount);

        if (deviceCount == 0) {
            return;
        }

        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, 0);

        warpSize = deviceProp.warpSize;
    }
}

void CUDA::PrintInfo() {
    if (deviceCount == -1) {
        GetProperty();
    }

    printf("CUDA: deviceCount: %i\n", deviceCount);

    if (deviceCount == 0) {
        return;
    }

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    printf("CUDA: warpSize: %i\n", deviceProp.warpSize);
}

namespace {
    void GetForceCPU(int count, int offset, float* masses, float* positionsX, float* positionsY, float* forcesX, float* forcesY, int threadId) {
        double _constGravity = 0.01f;
        int statIndex = threadId * offset;
        int endIndex = statIndex + offset;
        if (endIndex > count) {
            endIndex = count;
        }

        int sizeData = count;

        for (int index = statIndex; index < endIndex; ++index) {
            for (size_t otherIndex = 0; otherIndex < sizeData; ++otherIndex) {
                if (index == otherIndex) {
                    continue;
                }

                float gravityVecX = positionsX[otherIndex] - positionsX[index];
                float gravityVecY = positionsY[otherIndex] - positionsY[index];

                double dist = sqrt(gravityVecX * gravityVecX + gravityVecY * gravityVecY);
                gravityVecX /= dist;
                gravityVecY /= dist;

                double force = _constGravity * (masses[index] * masses[otherIndex]) / (dist * dist);
                gravityVecX *= force;
                gravityVecY *= force;

                forcesX[index] += gravityVecX;
                forcesY[index] += gravityVecY;
            }
        }
    }
}

__global__
void GetForceGPU(int* count, int* offset, float* masses, float* positionsX, float* positionsY, float* forcesX, float* forcesY) {
    double _constGravity = 0.01f;
    int statIndex = blockIdx.x * *offset;
    int endIndex = statIndex + *offset;
    if (endIndex > *count) {
        endIndex = *count;
    }

    int sizeData = *count;

    for (int index = statIndex; index < endIndex; ++index) {
        for (size_t otherIndex = 0; otherIndex < sizeData; ++otherIndex) {
            if (index == otherIndex) {
                continue;
            }

            float gravityVecX = positionsX[otherIndex] - positionsX[index];
            float gravityVecY = positionsY[otherIndex] - positionsY[index];

            double dist = sqrt(gravityVecX * gravityVecX + gravityVecY * gravityVecY);
            gravityVecX /= dist;
            gravityVecY /= dist;

            double force = _constGravity * (masses[index] * masses[otherIndex]) / (dist * dist);
            gravityVecX *= force;
            gravityVecY *= force;

            forcesX[index] += gravityVecX;
            forcesY[index] += gravityVecY;
        }
    }
}

void CUDA::GetForcesStaticTest(int count, float* masses, float* positionsX, float* positionsY, float* forcesX, float* forcesY) {
    unsigned int counThread = static_cast<double>(std::thread::hardware_concurrency());
    unsigned int sizeB = count;

    if ((sizeB * 2) > counThread) {
        int offst = sizeB / counThread;
        if ((sizeB % counThread) > 0) {
            ++offst;
        }

        std::vector<std::thread> threads;
        threads.reserve(counThread);

        for (unsigned int threadId = 0; threadId < counThread; ++threadId) {
            threads.emplace_back([&]() {
                GetForceCPU(count, offst, masses, positionsX, positionsY, forcesX, forcesY, threadId);
            });
        }

        for (std::thread& th : threads) {
            th.join();
        }
    }
    else
    {
        GetForceCPU(count, count, masses, positionsX, positionsY, forcesX, forcesY, 0);
    }
}

void CUDA::GetForcesStatic(int count, float* masses, float* positionsX, float* positionsY, float* forcesX, float* forcesY) {
    unsigned int counThread = CUDA::warpSize;
    if (counThread == 0) {
        return;
    }

    int offst = count / counThread;
    if ((count % counThread) > 0) {
        ++offst;
    }

    //...
    int* devCount;
    int* devOffset;
    float* devMasses;
    float* devPositionsX;
    float* devPositionsY;
    float* devForcesX;
    float* devForcesY;

    hipMalloc(&devCount,               sizeof(float));
    hipMalloc(&devOffset,              sizeof(float));
    hipMalloc(&devMasses,      count * sizeof(float));
    hipMalloc(&devPositionsX,  count * sizeof(float));
    hipMalloc(&devPositionsY,  count * sizeof(float));
    hipMalloc(&devForcesX,     count * sizeof(float));
    hipMalloc(&devForcesY,     count * sizeof(float));

    hipMemcpy(devCount,        &count,     count * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devOffset,       &offst,     count * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devMasses,       masses,     count * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devPositionsX,   positionsX, count * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devPositionsY,   positionsY, count * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devForcesX,      forcesX,    count * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devForcesY,      forcesY,    count * sizeof(float), hipMemcpyHostToDevice);

    GetForceGPU <<<counThread, 1>>> (devCount, devOffset, devMasses, devPositionsX, devPositionsY, devForcesX, devForcesY);

    hipMemcpy(devForcesX, forcesX, count * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(devForcesY, forcesY, count * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(devCount);
    hipFree(devOffset);
    hipFree(devMasses);
    hipFree(positionsX);
    hipFree(positionsY);
    hipFree(devForcesX);
    hipFree(devForcesY);
}

//...
#else
    void testCUDA(void) {}

    void CUDA::GetProperty() {}
    void CUDA::PrintInfo() {}
#endif
