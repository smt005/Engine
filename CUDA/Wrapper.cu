#include "hip/hip_runtime.h"

#include "Wrapper.h"
#include <thread>
#include <vector>

std::string CUDA::nameGPU;
int         CUDA::deviceCount = -1;
int         CUDA::warpSize = 0;
int         CUDA::maxThreadsPerBlock = 0;
int         CUDA::maxThreadsDim[3];
int         CUDA::maxGridSize[3];
int         CUDA::maxThreadsPerMultiProcessor = 0;
int         CUDA::maxBlocksPerMultiProcessor = 0;

#if ENABLE_CUDA

#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>

void CUDA::GetProperty() {
    if (deviceCount == -1) {
        hipGetDeviceCount(&deviceCount);

        if (deviceCount == 0) {
            return;
        }

        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, 0);

        nameGPU = deviceProp.name;
        warpSize = deviceProp.warpSize;
        maxThreadsPerBlock = deviceProp.maxThreadsPerBlock;
        maxThreadsDim[0] = deviceProp.maxThreadsDim[0];
        maxThreadsDim[1] = deviceProp.maxThreadsDim[1];
        maxThreadsDim[2] = deviceProp.maxThreadsDim[2];
        maxThreadsDim[3] = deviceProp.maxThreadsDim[3];
        maxGridSize[0] = deviceProp.maxGridSize[0];
        maxGridSize[1] = deviceProp.maxGridSize[1];
        maxGridSize[2] = deviceProp.maxGridSize[2];
        maxGridSize[3] = deviceProp.maxGridSize[3];
        maxThreadsPerMultiProcessor = deviceProp.maxThreadsPerMultiProcessor;
        maxBlocksPerMultiProcessor = deviceProp.maxBlocksPerMultiProcessor;
    }
}

void CUDA::PrintInfo() {
    if (deviceCount == -1) {
        GetProperty();
    }

    printf("CUDA: deviceCount: %i\n", deviceCount);

    if (deviceCount == 0) {
        return;
    }

    printf("CUDA:                     nameGPU: %s\n", nameGPU.c_str());
    printf("CUDA:                    warpSize: %i\n", warpSize);
    printf("CUDA:          maxThreadsPerBlock: %i\n", maxThreadsPerBlock);
    printf("CUDA:            maxThreadsDim[0]: %i\n", maxThreadsDim[0]);
    printf("CUDA:            maxThreadsDim[1]: %i\n", maxThreadsDim[1]);
    printf("CUDA:            maxThreadsDim[2]: %i\n", maxThreadsDim[2]);
    printf("CUDA:            maxThreadsDim[3]: %i\n", maxThreadsDim[3]);
    printf("CUDA:              maxGridSize[0]: %i\n", maxGridSize[0]);
    printf("CUDA:              maxGridSize[1]: %i\n", maxGridSize[1]);
    printf("CUDA:              maxGridSize[2]: %i\n", maxGridSize[2]);
    printf("CUDA:              maxGridSize[3]: %i\n", maxGridSize[3]);
    printf("CUDA: maxThreadsPerMultiProcessor: %i\n", maxThreadsPerMultiProcessor);
    printf("CUDA:  maxBlocksPerMultiProcessor: %i\n", maxBlocksPerMultiProcessor);
}

namespace {
    void GetForceCPU(int count, int offset, float* masses, float* positionsX, float* positionsY, float* forcesX, float* forcesY, int threadId) {
        double _constGravity = 0.01f;
        int statIndex = threadId * offset;
        int endIndex = statIndex + offset;
        if (endIndex > count) {
            endIndex = count;
        }

        int sizeData = count;

        for (int index = statIndex; index < endIndex; ++index) {
            for (size_t otherIndex = 0; otherIndex < sizeData; ++otherIndex) {
                if (index == otherIndex) {
                    continue;
                }

                float gravityVecX = positionsX[otherIndex] - positionsX[index];
                float gravityVecY = positionsY[otherIndex] - positionsY[index];

                double dist = sqrt(gravityVecX * gravityVecX + gravityVecY * gravityVecY);
                gravityVecX /= dist;
                gravityVecY /= dist;

                double force = _constGravity * (masses[index] * masses[otherIndex]) / (dist * dist);
                gravityVecX *= force;
                gravityVecY *= force;

                forcesX[index] += gravityVecX;
                forcesY[index] += gravityVecY;
            }
        }
    }
}

__global__
void GetForceGPU(int* count, int* offset, float* masses, float* positionsX, float* positionsY, float* forcesX, float* forcesY) {
    double _constGravity = 0.01f;
    int statIndex = blockIdx.x * *offset;
    int endIndex = statIndex + *offset;
    if (endIndex > *count) {
        endIndex = *count;
    }

    int sizeData = *count;

    for (int index = statIndex; index < endIndex; ++index) {
        for (size_t otherIndex = 0; otherIndex < sizeData; ++otherIndex) {
            if (index == otherIndex) {
                continue;
            }

            float gravityVecX = positionsX[otherIndex] - positionsX[index];
            float gravityVecY = positionsY[otherIndex] - positionsY[index];

            double dist = sqrt(gravityVecX * gravityVecX + gravityVecY * gravityVecY);
            gravityVecX /= dist;
            gravityVecY /= dist;

            double force = _constGravity * (masses[index] * masses[otherIndex]) / (dist * dist);
            gravityVecX *= force;
            gravityVecY *= force;

            forcesX[index] += gravityVecX;
            forcesY[index] += gravityVecY;
        }
    }
}

void CUDA::GetForcesStaticTest(int count, float* masses, float* positionsX, float* positionsY, float* forcesX, float* forcesY) {
    unsigned int counThread = static_cast<double>(std::thread::hardware_concurrency());
    unsigned int sizeB = count;

    if ((sizeB * 2) > counThread) {
        int offst = sizeB / counThread;
        if ((sizeB % counThread) > 0) {
            ++offst;
        }

        std::vector<std::thread> threads;
        threads.reserve(counThread);

        for (unsigned int threadId = 0; threadId < counThread; ++threadId) {
            threads.emplace_back([&]() {
                GetForceCPU(count, offst, masses, positionsX, positionsY, forcesX, forcesY, threadId);
            });
        }

        for (std::thread& th : threads) {
            th.join();
        }
    }
    else
    {
        GetForceCPU(count, count, masses, positionsX, positionsY, forcesX, forcesY, 0);
    }
}

void CUDA::GetForcesStatic(int count, float* masses, float* positionsX, float* positionsY, float* forcesX, float* forcesY) {
    unsigned int counThread = CUDA::warpSize;
    if (counThread == 0) {
        return;
    }

    int offst = count / counThread;
    if ((count % counThread) > 0) {
        ++offst;
    }

    //...
    int* devCount;
    int* devOffset;
    float* devMasses;
    float* devPositionsX;
    float* devPositionsY;
    float* devForcesX;
    float* devForcesY;

    hipMalloc(&devCount,               sizeof(float));
    hipMalloc(&devOffset,              sizeof(float));
    hipMalloc(&devMasses,      count * sizeof(float));
    hipMalloc(&devPositionsX,  count * sizeof(float));
    hipMalloc(&devPositionsY,  count * sizeof(float));
    hipMalloc(&devForcesX,     count * sizeof(float));
    hipMalloc(&devForcesY,     count * sizeof(float));

    hipMemcpy(devCount,        &count,     count * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devOffset,       &offst,     count * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devMasses,       masses,     count * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devPositionsX,   positionsX, count * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devPositionsY,   positionsY, count * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devForcesX,      forcesX,    count * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devForcesY,      forcesY,    count * sizeof(float), hipMemcpyHostToDevice);

    GetForceGPU <<<counThread, 1>>> (devCount, devOffset, devMasses, devPositionsX, devPositionsY, devForcesX, devForcesY);

    hipMemcpy(devForcesX, forcesX, count * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(devForcesY, forcesY, count * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(devCount);
    hipFree(devOffset);
    hipFree(devMasses);
    hipFree(positionsX);
    hipFree(positionsY);
    hipFree(devForcesX);
    hipFree(devForcesY);
}

//...
#else
    void testCUDA(void) {}

    void CUDA::GetProperty() {}
    void CUDA::PrintInfo() {}
#endif
