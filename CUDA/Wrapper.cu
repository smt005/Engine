#include "hip/hip_runtime.h"

#include "Wrapper.h"
#include <thread>
#include <vector>

int CUDA::deviceCount = -1;
int CUDA::warpSize = 0;

#if ENABLE_CUDA

#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>

__global__
void saxpy(int n, float a, float* x, float* y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) y[i] = a * x[i] + y[i];
}

void testCUDA(void)
{
    int N = 1 << 20;
    float* x, * y, * d_x, * d_y;
    x = (float*)malloc(N * sizeof(float));
    y = (float*)malloc(N * sizeof(float));

    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));

    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

    // Perform SAXPY on 1M elements
    saxpy << <(N + 255) / 256, 256 >> > (N, 2.0f, d_x, d_y);

    hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = std::max(maxError, std::abs(y[i] - 4.0f));
    printf("Max error: %f\n", maxError);

    hipFree(d_x);
    hipFree(d_y);
    free(x);
    free(y);
}

void CUDA::GetProperty() {
    if (deviceCount == -1) {
        hipGetDeviceCount(&deviceCount);
    }
}

void CUDA::PrintInfo() {
    if (deviceCount == -1) {
        GetProperty();
    }

    printf("CUDA: deviceCount: %i\n", deviceCount);

    if (deviceCount == 0) {
        return;
    }

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    printf("CUDA: warpSize: %i\n", deviceProp.warpSize);
}

namespace {
    void GetForce(int count, int offset, float* masses, float* positionsX, float* positionsY, float* forcesX, float* forcesY, int threadId) {
        double _constGravity = 0.01f;
        int statIndex = threadId * offset;
        int endIndex = statIndex + offset;
        if (endIndex > count) {
            endIndex = count;
        }

        int sizeData = count;

        for (int index = statIndex; index < endIndex; ++index) {
            for (size_t otherIndex = 0; otherIndex < sizeData; ++otherIndex) {
                if (index == otherIndex) {
                    continue;
                }

                float gravityVecX = positionsX[otherIndex] - positionsX[index];
                float gravityVecY = positionsY[otherIndex] - positionsY[index];

                double dist = sqrt(gravityVecX * gravityVecX + gravityVecY * gravityVecY);
                gravityVecX /= dist;
                gravityVecY /= dist;

                double force = _constGravity * (masses[index] * masses[otherIndex]) / (dist * dist);
                gravityVecX *= force;
                gravityVecY *= force;

                forcesX[index] += gravityVecX;
                forcesY[index] += gravityVecY;
            }
        }
    }
}

void CUDA::GetForcesStaticTest(int count, float* masses, float* positionsX, float* positionsY, float* forcesX, float* forcesY) {
    unsigned int counThread = static_cast<double>(std::thread::hardware_concurrency());
    unsigned int sizeB = count;

    if ((sizeB * 2) > counThread) {
        int offst = sizeB / counThread;
        if ((sizeB % counThread) > 0) {
            ++offst;
        }

        std::vector<std::thread> threads;
        threads.reserve(counThread);

        for (unsigned int threadId = 0; threadId < counThread; ++threadId) {
            threads.emplace_back([&]() {
                GetForce(count, offst, masses, positionsX, positionsY, forcesX, forcesY, threadId);
            });
        }

        for (std::thread& th : threads) {
            th.join();
        }
    }
    else
    {
        GetForce(count, count, masses, positionsX, positionsY, forcesX, forcesY, 0);
    }
}

//...
#else
    void testCUDA(void) {}

    void CUDA::GetProperty() {}
    void CUDA::PrintInfo() {}
#endif
